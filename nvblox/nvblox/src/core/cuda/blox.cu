#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/core/blox.h"
#include "nvblox/core/common_names.h"

namespace nvblox {

// Must be called with:
// - a single block
// - one thread per voxel
__global__ void setColorBlockGray(ColorBlock* block_device_ptr) {
  ColorVoxel* voxel_ptr =
      &block_device_ptr->voxels[threadIdx.z][threadIdx.y][threadIdx.x];
  voxel_ptr->color.r = 127;
  voxel_ptr->color.g = 127;
  voxel_ptr->color.b = 127;
  voxel_ptr->weight = 0.0f;
}

void setColorBlockGrayOnGPU(ColorBlock* block_device_ptr) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  setColorBlockGray<<<1, kThreadsPerBlock>>>(block_device_ptr);
  // NOTE(alexmillane): At the moment we launch this allocation on the default
  // stream which implicitly synchronizes. At some point in the future we should
  // probably move this to a stream.
  // checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipPeekAtLastError());
}

}  // namespace nvblox
