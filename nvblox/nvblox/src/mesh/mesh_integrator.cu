#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hip/hip_runtime.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "nvblox/core/accessors.h"
#include "nvblox/core/common_names.h"
#include "nvblox/integrators/integrators_common.h"
#include "nvblox/mesh/impl/marching_cubes_table.h"
#include "nvblox/mesh/marching_cubes.h"
#include "nvblox/mesh/mesh_integrator.h"
#include "nvblox/utils/timing.h"

namespace nvblox {

MeshIntegrator::~MeshIntegrator() {
  if (cuda_stream_ != nullptr) {
    hipStreamDestroy(cuda_stream_);
  }
}

bool MeshIntegrator::integrateBlocksGPU(
    const TsdfLayer& distance_layer, const std::vector<Index3D>& block_indices,
    BlockLayer<MeshBlock>* mesh_layer) {
  timing::Timer mesh_timer("mesh/gpu/integrate");
  CHECK_NOTNULL(mesh_layer);
  CHECK_NEAR(distance_layer.block_size(), mesh_layer->block_size(), 1e-4);
  if (block_indices.empty()) {
    return true;
  }

  // Initialize the stream if not done yet.
  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  // Figure out which of these actually contain something worth meshing.
  float block_size = distance_layer.block_size();
  float voxel_size = distance_layer.voxel_size();

  // Clear all blocks if they exist.
  for (const Index3D& block_index : block_indices) {
    MeshBlock::Ptr mesh_block = mesh_layer->getBlockAtIndex(block_index);
    if (mesh_block) {
      mesh_block->clear();
    }
  }

  // First create a list of meshable blocks.
  std::vector<Index3D> meshable_blocks;
  timing::Timer meshable_timer("mesh/gpu/get_meshable");
  getMeshableBlocksGPU(distance_layer, block_indices, 5 * voxel_size,
                       &meshable_blocks);
  meshable_timer.Stop();

  // Then get all the candidates and mesh each block.
  timing::Timer mesh_blocks_timer("mesh/gpu/mesh_blocks");

  meshBlocksGPU(distance_layer, meshable_blocks, mesh_layer);

  // TODO: optionally weld here as well.
  mesh_blocks_timer.Stop();

  return true;
}

// Kernels

// Takes in a vector of blocks, and outputs an integer true if that block is
// meshable.
// Block size MUST be voxels_per_side x voxels_per_side x voxel_per_size.
// Grid size can be anything.
__global__ void isBlockMeshableKernel(int num_blocks,
                                      const VoxelBlock<TsdfVoxel>** blocks,
                                      float cutoff_distance, float min_weight,
                                      bool* meshable) {
  dim3 voxel_index = threadIdx;
  // This for loop allows us to have fewer threadblocks than there are
  // blocks in this computation. We assume the threadblock size is constant
  // though to make our lives easier.
  for (int block_index = blockIdx.x; block_index < num_blocks;
       block_index += gridDim.x) {
    // Get the correct voxel for this index.
    const TsdfVoxel& voxel =
        blocks[block_index]
            ->voxels[voxel_index.z][voxel_index.y][voxel_index.x];
    if (fabs(voxel.distance) <= cutoff_distance && voxel.weight >= min_weight) {
      meshable[block_index] = true;
    }
  }
}

// Takes in a set of blocks arranged in neighbor sets and their relative
// positions, then finds vertex candidates, and finally creates the output
// meshes for them.
// Block size MUST be voxels_per_side x voxels_per_side x voxel_per_size.
// Grid size can be anything.
__global__ void meshBlocksCalculateTableIndicesKernel(
    int num_blocks, const VoxelBlock<TsdfVoxel>** blocks,
    const Vector3f* block_positions, float voxel_size, float min_weight,
    marching_cubes::PerVoxelMarchingCubesResults* marching_cubes_results,
    int* mesh_block_sizes) {
  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;
  constexpr int kVoxelsPerBlock =
      kVoxelsPerSide * kVoxelsPerSide * kVoxelsPerSide;
  constexpr int kCubeNeighbors = 8;

  const dim3 voxel_index = dim3(threadIdx.z, threadIdx.y, threadIdx.x);

  const int linear_thread_idx =
      threadIdx.x +
      kVoxelsPerSide * (threadIdx.y + kVoxelsPerSide * threadIdx.z);

  // Preallocate a half voxel size.
  const Vector3f half_voxel(0.5f, 0.5f, 0.5f);

  marching_cubes::PerVoxelMarchingCubesResults marching_cubes_results_local;

  // This for loop allows us to have fewer threadblocks than there are
  // blocks in this computation. We assume the threadblock size is constant
  // though to make our lives easier.
  for (int block_index = blockIdx.x; block_index < num_blocks;
       block_index += gridDim.x) {
    // Initialize the calculated output size for this block.
    __shared__ int mesh_block_size;
    if (linear_thread_idx == 0) {
      mesh_block_size = 0;
    }
    __syncthreads();

    // Getting the block pointer is complicated now so let's just get it.
    const VoxelBlock<TsdfVoxel>* block = blocks[block_index * kCubeNeighbors];

    // Get the linear index of the this voxel in this block
    const int vertex_neighbor_idx =
        block_index * kVoxelsPerBlock + linear_thread_idx;

    // Check all 8 neighbors.
    bool skip_voxel = false;
    for (unsigned int i = 0; i < 8; ++i) {
      Index3D corner_index(
          voxel_index.x + marching_cubes::kCornerIndexOffsets[i][0],
          voxel_index.y + marching_cubes::kCornerIndexOffsets[i][1],
          voxel_index.z + marching_cubes::kCornerIndexOffsets[i][2]);
      Index3D block_offset(0, 0, 0);
      bool search_neighbor = false;
      // Are we in bounds? If not, have to get a neighbor.
      // The neighbor should correspond to the index in neighbor blocks.
      for (int j = 0; j < 3; j++) {
        if (corner_index[j] >= kVoxelsPerSide) {
          // Here the index is too much.
          corner_index(j) -= kVoxelsPerSide;
          block_offset(j) = 1;
          search_neighbor = true;
        }
      }

      const TsdfVoxel* voxel = nullptr;
      // Don't look for neighbors for now.
      if (search_neighbor) {
        int neighbor_index =
            marching_cubes::neighborIndexFromDirection(block_offset);
        const VoxelBlock<TsdfVoxel>* neighbor_block =
            blocks[block_index * kCubeNeighbors + neighbor_index];
        if (neighbor_block == nullptr) {
          skip_voxel = true;
          break;
        }
        voxel =
            &neighbor_block
                 ->voxels[corner_index.x()][corner_index.y()][corner_index.z()];
      } else {
        voxel =
            &block
                 ->voxels[corner_index.x()][corner_index.y()][corner_index.z()];
      }
      // If any of the neighbors are not observed, this can't be a mesh
      // triangle.
      if (voxel->weight < min_weight) {
        skip_voxel = true;
        break;
      }

      // Calculate the position of this voxel.
      marching_cubes_results_local.vertex_sdf[i] = voxel->distance;
      marching_cubes_results_local.vertex_coords[i] =
          block_positions[block_index] +
          voxel_size * (corner_index.cast<float>() + half_voxel +
                        (kVoxelsPerSide * block_offset).cast<float>());
    }

    if (!skip_voxel) {
      // If we've made it this far, this needs to be meshed.
      marching_cubes_results_local.contains_mesh = true;

      // Calculate the index into the magic marching cubes table
      marching_cubes_results_local.marching_cubes_table_index =
          marching_cubes::calculateVertexConfiguration(
              marching_cubes_results_local.vertex_sdf);

      // Mesh this cube. This will keep track of what index we're at within
      // the cube.
      marching_cubes::calculateOutputIndex(&marching_cubes_results_local,
                                           &mesh_block_size);

      // Write out to global memory
      marching_cubes_results[vertex_neighbor_idx] =
          marching_cubes_results_local;
    }

    // Writing the shared variable block size to global memory (per block)
    __syncthreads();
    if (linear_thread_idx == 0) {
      mesh_block_sizes[block_index] = mesh_block_size;
    }
  }
}

__global__ void meshBlocksCalculateVerticesKernel(
    int num_blocks,
    const marching_cubes::PerVoxelMarchingCubesResults* marching_cubes_results,
    const int* mesh_block_sizes, CudaMeshBlock* mesh_blocks) {
  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;

  const int linear_thread_idx =
      threadIdx.x +
      kVoxelsPerSide * (threadIdx.y + kVoxelsPerSide * threadIdx.z);

  // This for loop allows us to have fewer threadblocks than there are
  // blocks in this computation. We assume the threadblock size is constant
  // though to make our lives easier.
  for (int block_index = blockIdx.x; block_index < num_blocks;
       block_index += gridDim.x) {
    // If this block contains a mesh
    if (mesh_block_sizes[block_index] > 0) {
      // Get the linear index of the this voxel in this block
      constexpr int kVoxelsPerBlock =
          kVoxelsPerSide * kVoxelsPerSide * kVoxelsPerSide;
      const int vertex_neighbor_idx =
          block_index * kVoxelsPerBlock + linear_thread_idx;

      // If this voxel contains a mesh
      if (marching_cubes_results[vertex_neighbor_idx].contains_mesh) {
        // Convert the marching cube table index into vertex coordinates
        marching_cubes::calculateVertices(
            marching_cubes_results[vertex_neighbor_idx],
            &mesh_blocks[block_index]);
      }
    }
  }
}

// Wrappers

void MeshIntegrator::getMeshableBlocksGPU(
    const TsdfLayer& distance_layer, const std::vector<Index3D>& block_indices,
    float cutoff_distance, std::vector<Index3D>* meshable_blocks) {
  CHECK_NOTNULL(meshable_blocks);
  if (block_indices.size() == 0) {
    return;
  }

  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;
  // One block per block, 1 thread per pixel. :)
  // Dim block can be smaller, but dim_threads must be the same.
  int dim_block = block_indices.size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);

  // Collect all the meshable blocks as raw pointers.
  // Get all the block pointers and positions.
  block_ptrs_host_.resize(block_indices.size());

  for (size_t i = 0; i < block_indices.size(); i++) {
    block_ptrs_host_[i] =
        distance_layer.getBlockAtIndex(block_indices[i]).get();
  }

  block_ptrs_device_ = block_ptrs_host_;

  // Allocate a device vector that holds the meshable result.
  meshable_device_.resize(block_indices.size());
  meshable_device_.setZero();

  checkCudaErrors(hipPeekAtLastError());
  isBlockMeshableKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      block_indices.size(), block_ptrs_device_.data(), cutoff_distance,
      min_weight_, meshable_device_.data());
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  meshable_host_ = meshable_device_;

  for (size_t i = 0; i < block_indices.size(); i++) {
    if (meshable_host_[i]) {
      meshable_blocks->push_back(block_indices[i]);
    }
  }
}

void MeshIntegrator::meshBlocksGPU(const TsdfLayer& distance_layer,
                                   const std::vector<Index3D>& block_indices,
                                   BlockLayer<MeshBlock>* mesh_layer) {
  if (block_indices.empty()) {
    return;
  }
  timing::Timer mesh_prep_timer("mesh/gpu/mesh_blocks/prep");
  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;
  constexpr int kCubeNeighbors = 8;

  // One block per block, 1 thread per voxel. :)
  // Dim block can be smaller, but dim_threads must be the same.
  int dim_block = block_indices.size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);

  // Get the block and voxel size.
  const float block_size = distance_layer.block_size();
  const float voxel_size = distance_layer.voxel_size();

  // Get all the block pointers and positions.
  // Block pointers are actually a 2D array of also the neighbor block pointers
  // The neighbors CAN be null so they need to be checked.
  block_ptrs_host_.resize(block_indices.size() * kCubeNeighbors);
  block_positions_host_.resize(block_indices.size());
  for (size_t i = 0; i < block_indices.size(); i++) {
    block_ptrs_host_[i * kCubeNeighbors] =
        distance_layer.getBlockAtIndex(block_indices[i]).get();
    for (size_t j = 1; j < kCubeNeighbors; j++) {
      // Get the pointers to all the neighbors as well.
      block_ptrs_host_[i * kCubeNeighbors + j] =
          distance_layer
              .getBlockAtIndex(block_indices[i] +
                               marching_cubes::directionFromNeighborIndex(j))
              .get();
    }
    block_positions_host_[i] =
        getPositionFromBlockIndex(block_size, block_indices[i]);
  }

  // Create an output mesh blocks vector..
  mesh_blocks_host_.resize(block_indices.size());

  block_ptrs_device_ = block_ptrs_host_;
  block_positions_device_ = block_positions_host_;

  // Allocate working space
  constexpr int kNumVoxelsPerBlock =
      kVoxelsPerSide * kVoxelsPerSide * kVoxelsPerSide;
  marching_cubes_results_device_.resize(block_indices.size() *
                                        kNumVoxelsPerBlock);
  marching_cubes_results_device_.setZero();
  mesh_block_sizes_device_.resize(block_indices.size());
  mesh_block_sizes_device_.setZero();
  mesh_prep_timer.Stop();

  // Run the first half of marching cubes and calculate:
  // - the per-vertex indexes into the magic triangle table
  // - the number of vertices in each mesh block.
  timing::Timer mesh_kernel_1_timer("mesh/gpu/mesh_blocks/kernel_table");
  meshBlocksCalculateTableIndicesKernel<<<dim_block, dim_threads, 0,
                                          cuda_stream_>>>(
      block_indices.size(), block_ptrs_device_.data(),
      block_positions_device_.data(), voxel_size, min_weight_,
      marching_cubes_results_device_.data(), mesh_block_sizes_device_.data());
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  mesh_kernel_1_timer.Stop();

  // Copy back the new mesh block sizes (so we can allocate space)
  timing::Timer mesh_copy_timer("mesh/gpu/mesh_blocks/copy_out");
  mesh_block_sizes_host_ = mesh_block_sizes_device_;
  mesh_copy_timer.Stop();

  // Allocate mesh blocks
  timing::Timer mesh_allocation_timer("mesh/gpu/mesh_blocks/block_allocation");
  for (size_t i = 0; i < block_indices.size(); i++) {
    const int num_vertices = mesh_block_sizes_host_[i];

    if (num_vertices > 0) {
      MeshBlock::Ptr output_block =
          mesh_layer->allocateBlockAtIndex(block_indices[i]);

      // Grow the vector with a growth factor and a minimum allocation to avoid
      // repeated reallocation
      if (num_vertices > output_block->capacity()) {
        constexpr int kMinimumMeshBlockTrianglesPerVoxel = 1;
        constexpr int kMinimumMeshBlockVertices =
            kNumVoxelsPerBlock * kMinimumMeshBlockTrianglesPerVoxel * 3;
        constexpr int kMeshBlockOverallocationFactor = 2;
        const int num_vertices_to_allocate =
            std::max(kMinimumMeshBlockVertices,
                     num_vertices * kMeshBlockOverallocationFactor);
        output_block->reserveNumberOfVertices(num_vertices_to_allocate);
      }
      output_block->resizeToNumberOfVertices(num_vertices);
      mesh_blocks_host_[i] = CudaMeshBlock(output_block.get());
    }
  }
  mesh_blocks_device_ = mesh_blocks_host_;
  mesh_allocation_timer.Stop();

  // Run the second half of marching cubes
  // - Translating the magic table indices into triangle vertices and writing
  //   them into the mesh layer.
  timing::Timer mesh_kernel_2_timer("mesh/gpu/mesh_blocks/kernel_vertices");
  meshBlocksCalculateVerticesKernel<<<dim_block, dim_threads, 0,
                                      cuda_stream_>>>(
      block_indices.size(), marching_cubes_results_device_.data(),
      mesh_block_sizes_device_.data(), mesh_blocks_device_.data());
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  mesh_kernel_2_timer.Stop();

  // Optional third stage: welding.
  if (weld_vertices_) {
    timing::Timer welding_timer("mesh/gpu/mesh_blocks/welding");
    weldVertices(block_indices, mesh_layer);
  }
}

void MeshIntegrator::weldVertices(const std::vector<Index3D>& block_indices,
                                  BlockLayer<MeshBlock>* mesh_layer) {
  for (const Index3D& index : block_indices) {
    MeshBlock::Ptr mesh_block = mesh_layer->getBlockAtIndex(index);

    if (!mesh_block || mesh_block->size() <= 3) {
      continue;
    }

    // Store a copy of the input vertices.
    input_vertices_ = mesh_block->vertices;
    input_normals_ = mesh_block->normals;

    // sort vertices to bring duplicates together
    thrust::sort(thrust::device, mesh_block->vertices.begin(),
                 mesh_block->vertices.end(), VectorCompare<Vector3f>());

    // Find unique vertices and erase redundancies. The iterator will point to
    // the new last index.
    auto iterator = thrust::unique(thrust::device, mesh_block->vertices.begin(),
                                   mesh_block->vertices.end());

    // Figure out the new size.
    size_t new_size = iterator - mesh_block->vertices.begin();
    mesh_block->vertices.resize(new_size);
    mesh_block->normals.resize(new_size);

    // Find the indices of the original triangles.
    thrust::lower_bound(thrust::device, mesh_block->vertices.begin(),
                        mesh_block->vertices.end(), input_vertices_.begin(),
                        input_vertices_.end(), mesh_block->triangles.begin(),
                        VectorCompare<Vector3f>());

    // Reshuffle the normals to match.
    thrust::scatter(thrust::device, input_normals_.begin(),
                    input_normals_.end(), mesh_block->triangles.begin(),
                    mesh_block->normals.begin());
  }
}

}  // namespace nvblox