/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/experiments/integrators/cuda/depth_frame_texture.cuh"

#include "nvblox/core/cuda/error_check.cuh"

namespace nvblox {
namespace experiments {

DepthImageTexture::DepthImageTexture(const DepthImage& depth_frame,
                                     hipStream_t transfer_stream) {
  // Note(alexmillane): Taken from texture memory example
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-memory

  // Allocate CUDA array in device memory
  // Each channel is a 32bit float in the first (x) dimension
  const hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  checkCudaErrors(hipMallocArray(&depth_array_, &channelDesc,
                                  depth_frame.width(), depth_frame.height()));

  // Set pitch of the source (the width in memory in bytes of the 2D array
  // pointed to by src, including padding), we dont have any padding
  const size_t spitch = depth_frame.width() * sizeof(float);
  // Copy data located at address h_data in host memory to device memory
  checkCudaErrors(hipMemcpy2DToArrayAsync(
      depth_array_, 0, 0, depth_frame.dataConstPtr(), spitch,
      depth_frame.width() * sizeof(float), depth_frame.height(),
      hipMemcpyDefault, transfer_stream));

  // Specify texture
  struct hipResourceDesc resource_description;
  memset(&resource_description, 0, sizeof(resource_description));
  resource_description.resType = hipResourceTypeArray;
  resource_description.res.array.array = depth_array_;

  // Specify texture object parameters
  struct hipTextureDesc texture_description;
  memset(&texture_description, 0, sizeof(texture_description));
  texture_description.addressMode[0] = hipAddressModeClamp;
  texture_description.addressMode[1] = hipAddressModeClamp;
  texture_description.filterMode = hipFilterModeLinear;
  texture_description.readMode = hipReadModeElementType;
  texture_description.normalizedCoords = 0;

  // Create texture object
  checkCudaErrors(hipCreateTextureObject(
      &depth_texture_, &resource_description, &texture_description, NULL));
}

DepthImageTexture::~DepthImageTexture() {
  hipDestroyTextureObject(depth_texture_);
  hipFreeArray(depth_array_);
}

}  // namespace experiments
}  // namespace nvblox
