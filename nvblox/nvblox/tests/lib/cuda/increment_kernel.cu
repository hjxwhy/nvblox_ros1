#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/tests/increment_on_gpu.h"

__global__ void incrementKernel(int* number) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    (*number)++;
  }
}

__global__ void incrementKernel(int* number, const int num_elelments) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elelments) {
    number[idx]++;
  }
}


namespace nvblox {
namespace test_utils {

void incrementOnGPU(int* number) {
  incrementKernel<<<1, 1>>>(number);
  hipDeviceSynchronize();
}

void incrementOnGPU(const int num_elelments, int* number) {
  constexpr int kThreadsPerBlock = 32;
  const int num_blocks = (num_elelments / kThreadsPerBlock) + 1;
  incrementKernel<<<num_blocks, kThreadsPerBlock>>>(number, num_elelments);
  hipDeviceSynchronize();
}

}  // namespace test_utils
}  // namespace nvblox